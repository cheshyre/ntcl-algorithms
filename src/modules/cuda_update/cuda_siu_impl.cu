#include "hip/hip_runtime.h"
#include <complex>
#include <iostream>
#include <hip/hip_complex.h>

#include "cuda_common.h"

template <typename T>
__global__ void siu_kernel(T *dst, long nelements, T alpha) {
    long idx = ((long) blockIdx.x)*((long) blockDim.x) + threadIdx.x;

    if ( idx < nelements) dst[idx] = alpha*dst[idx];
}

__global__ void siu_kernel_c64(hipComplex *dst, long nelements, hipComplex alpha) {
    long idx = ((long) blockIdx.x)*((long) blockDim.x) + threadIdx.x;

    if ( idx < nelements) dst[idx] = hipCmulf(alpha,dst[idx]);
}

__global__ void siu_kernel_c128(hipDoubleComplex *dst, long nelements, hipDoubleComplex alpha) {
    long idx = ((long) blockIdx.x)*((long) blockDim.x) + threadIdx.x;

    if ( idx < nelements) dst[idx] = hipCmul(alpha,dst[idx]);
}

extern "C" void perform_siu_r32(float *dst, int64_t nelements, float alpha, hipStream_t *stream) {
    int block_size = 256;
    int nblocks = (int) (nelements-1)/(block_size)+1;

    dim3 blockGrid(nblocks);
    dim3 thread_per_block(block_size);
    if ( stream ) {
        siu_kernel<float><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
    } else {
        siu_kernel<float><<<nblocks, block_size>>>(dst, nelements, alpha);
    }
    check_cuda_error( hipPeekAtLastError() );
}

extern "C" void perform_siu_r64(double *dst, int64_t nelements, double alpha, hipStream_t *stream) {
    int block_size = 256;
    int nblocks = (int) (nelements-1)/(block_size)+1;

    dim3 blockGrid(nblocks);
    dim3 thread_per_block(block_size);
    if ( stream ) {
        siu_kernel<double><<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
    } else {
        siu_kernel<double><<<nblocks, block_size>>>(dst, nelements, alpha);
    }
    check_cuda_error( hipPeekAtLastError() );
}

extern "C" void perform_siu_c64(hipComplex *dst, int64_t nelements, hipComplex alpha, hipStream_t *stream) {
    int block_size = 256;
    int nblocks = (int) (nelements-1)/(block_size)+1;

    dim3 blockGrid(nblocks);
    dim3 thread_per_block(block_size);
    if ( stream ) {
        siu_kernel_c64<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
    } else {
        siu_kernel_c64<<<nblocks, block_size>>>(dst, nelements, alpha);
    }
    check_cuda_error( hipPeekAtLastError() );
}

extern "C" void perform_siu_c128(hipDoubleComplex *dst, int64_t nelements, hipDoubleComplex alpha, hipStream_t *stream) {
    int block_size = 256;
    int nblocks = (int) (nelements-1)/(block_size)+1;

    dim3 blockGrid(nblocks);
    dim3 thread_per_block(block_size);
    if ( stream ) {
        siu_kernel_c128<<<nblocks, block_size, 0, *stream>>>(dst, nelements, alpha);
    } else {
        siu_kernel_c128<<<nblocks, block_size>>>(dst, nelements, alpha);
    }
    check_cuda_error( hipPeekAtLastError() );
}
