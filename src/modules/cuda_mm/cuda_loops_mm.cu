#include "hip/hip_runtime.h"
#include <stdio.h>
#include <complex>
#include <hip/hip_complex.h>

#include "cuda_common.h"

#define BLOCKSIZE 32

template <typename T>
__global__ void mmp_double_kernel(T alpha, T beta, int m, int n, int k, T *a, T *b, T *c)
{
    double ctemp = 0.0;
    int row = blockIdx.x*blockDim.x + threadIdx.x; // Transposed from Fortran
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if ( row >=m || col >= n ) return;
    for (int i = 0; i < k; i++)
    {
        ctemp += a[i*m + row]*b[col*k + i];
    }
    c[col*m + row] = ctemp*alpha + c[col*m + row]*beta;
}

__global__ void mmp_c64(hipComplex alpha, hipComplex beta, int m, int n, int k, hipComplex *a, hipComplex *b, hipComplex *c)
{
    hipComplex ctemp;
    ctemp.x = 0.0;
    ctemp.y = 0.0;
    int row = blockIdx.x*blockDim.x + threadIdx.x; // Transposed from Fortran
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if ( row >=m || col >= n ) return;
    for (int i = 0; i < k; i++)
    {
        ctemp = hipCaddf(ctemp, hipCmulf(a[i*m + row], b[col*k + i]));
    }
    c[col*m + row] = hipCaddf(hipCmulf(ctemp, alpha), hipCmulf(c[col*m + row], beta));
} 

__global__ void mmp_c128(hipDoubleComplex alpha, hipDoubleComplex beta, int m, int n, int k, hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c)
{
    hipDoubleComplex ctemp;
    ctemp.x = 0.0;
    ctemp.y = 0.0;
    int row = blockIdx.x*blockDim.x + threadIdx.x; // Transposed from Fortran
    int col = blockIdx.y*blockDim.y + threadIdx.y;

    if ( row >=m || col >= n ) return;
    for (int i = 0; i < k; i++)
    {
        ctemp = hipCadd(ctemp, hipCmul(a[i*m + row], b[col*k + i]));
    }
    c[col*m + row] = hipCadd(hipCmul(ctemp, alpha), hipCmul(c[col*m + row], beta));
}


extern "C" int cuda_mmp_execute_r32(float alpha, float beta, int m, int n, int k,
                    float *a, float *b, float *c)
{
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    dim3 dimGrid((m-1)/(BLOCKSIZE)+1,(n-1)/(BLOCKSIZE)+1);
    mmp_double_kernel<<<dimGrid,dimBlock>>>(alpha, beta, m, n, k, a, b, c);
    return 0;
}

extern "C" int cuda_mmp_execute_r64(double alpha, double beta, int m, int n, int k,
                    double *a, double *b, double *c)
{
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    dim3 dimGrid((m-1)/(BLOCKSIZE)+1,(n-1)/(BLOCKSIZE)+1);
    mmp_double_kernel<<<dimGrid,dimBlock>>>(alpha, beta, m, n, k, a, b, c);
    return 0;
}

extern "C" int cuda_mmp_execute_c64(hipComplex *alpha, hipComplex *beta, int m, int n, int k,
                    hipComplex *a, hipComplex *b, hipComplex *c)
{
    // data was passed using pointer due to compiler bug. 
    //In the kernel function, to pass data from host to device, the address can not be used and 
    // need to be dereference by * to pass the a copy of the value
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    dim3 dimGrid((m-1)/(BLOCKSIZE)+1,(n-1)/(BLOCKSIZE)+1);
    mmp_c64<<<dimGrid,dimBlock>>>(*alpha, *beta, m, n, k, a, b, c);
    return 0;
}

extern "C" int cuda_mmp_execute_c128(hipDoubleComplex alpha, hipDoubleComplex beta, int m, int n, int k,
                    hipDoubleComplex *a, hipDoubleComplex *b, hipDoubleComplex *c)
{
    dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
    dim3 dimGrid((m-1)/(BLOCKSIZE)+1,(n-1)/(BLOCKSIZE)+1);
    mmp_c128<<<dimGrid,dimBlock>>>(alpha, beta, m, n, k, a, b, c);
    return 0;
}
