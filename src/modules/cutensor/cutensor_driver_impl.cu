#include <stdlib.h>
#include <iostream>
#include <complex>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

#include <unordered_map>
#include <vector>

// Handle cuTENSOR errors
#define HANDLE_ERROR(x) {                                                              \
  const auto err = x;                                                                  \
  if( x != HIPTENSOR_STATUS_SUCCESS )                                                   \
  { printf("Error: %s in line %d\n", hiptensorGetErrorString(x), __LINE__); exit(-1); } \
}

// Initialize cuTENSOR library
extern "C"
int cutensor_init(hiptensorHandle_t* &vhandle){
  hiptensorStatus_t err;

  vhandle = (hiptensorHandle_t*) malloc( sizeof(hiptensorHandle_t) );

  err = cutensorInit(vhandle);

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
    return -1;
  }

  return 0;
}

// free handle
extern "C"
int cutensor_free(hiptensorHandle_t* vhandle){

  free(vhandle);
  vhandle = NULL;

  return 0;
}

extern "C"
int cutensor_contract_r32(hiptensorHandle_t* handle,
    float* c, int c_rank, long* c_dims, int* c_inds,
    float* a, int a_rank, long* a_dims, int* a_inds,
    float* b, int b_rank, long* b_dims, int* b_inds,
    float alpha,
    float beta){

  hipDataType typeA = HIP_R_32F;
  hipDataType typeB = HIP_R_32F;
  hipDataType typeC = HIP_R_32F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

  /* ***************************** */

  // Create Tensor Descriptors
  hiptensorTensorDescriptor_t descA;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descA,
        a_rank,
        a_dims,
        NULL,/*stride*/
        typeA, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descB;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descB,
        b_rank,
        b_dims,
        NULL,/*stride*/
        typeB, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descC,
        c_rank,
        c_dims,
        NULL,/*stride*/
        typeC, HIPTENSOR_OP_IDENTITY ) );

  /* ***************************** */

  //Retrieve the memory alignment for each tensor
  uint32_t alignmentRequirementA;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        a,
        &descA,
        &alignmentRequirementA) );

  uint32_t alignmentRequirementB;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        b,
        &descB,
        &alignmentRequirementB) );

  uint32_t alignmentRequirementC;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        c,
        &descC,
        &alignmentRequirementC) );


  /* ***************************** */

  // Create the Contraction Descriptor
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( handle,
        &desc,
        &descA, a_inds, alignmentRequirementA,
        &descB, b_inds, alignmentRequirementB,
        &descC, c_inds, alignmentRequirementC,
        &descC, c_inds, alignmentRequirementC,
        typeCompute) );

  /* ***************************** */

  // Set the algorithm to use
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
        handle, &find,
        HIPTENSOR_ALGO_DEFAULT) );


  /* ***************************** */

  // Query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(handle,
        &desc,
        &find,
        CUTENSOR_WORKSPACE_RECOMMENDED, &worksize ) );

  // Allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) ) // This is optional!
    {
      work = nullptr;
      worksize = 0;
    }
  }


  /* ***************************** */

  // Create Contraction Plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(handle,
        &plan,
        &desc,
        &find,
        worksize) );

  /* ***************************** */

  hiptensorStatus_t err;

  // Execute the tensor contraction
  err = hiptensorContraction(handle,
      &plan,
      (void*)&alpha, a,
      b,
      (void*)&beta,  c,
      c,
      work, worksize, 0 /* stream */);
  /* hipDeviceSynchronize(); */

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  if ( work ) hipFree( work );

  return 0;
}

// init cutensor before use
extern "C"
int cutensor_contract_r64(hiptensorHandle_t* handle,
    double* c, int c_rank, long* c_dims, int* c_inds,
    double* a, int a_rank, long* a_dims, int* a_inds,
    double* b, int b_rank, long* b_dims, int* b_inds,
    double alpha,
    double beta){

  hipDataType typeA = HIP_R_64F;
  hipDataType typeB = HIP_R_64F;
  hipDataType typeC = HIP_R_64F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_64F;

  /* ***************************** */

  // Create Tensor Descriptors
  hiptensorTensorDescriptor_t descA;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descA,
        a_rank,
        a_dims,
        NULL,/*stride*/
        typeA, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descB;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descB,
        b_rank,
        b_dims,
        NULL,/*stride*/
        typeB, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descC,
        c_rank,
        c_dims,
        NULL,/*stride*/
        typeC, HIPTENSOR_OP_IDENTITY ) );

  /* ***************************** */

  //Retrieve the memory alignment for each tensor
  uint32_t alignmentRequirementA;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        a,
        &descA,
        &alignmentRequirementA) );

  uint32_t alignmentRequirementB;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        b,
        &descB,
        &alignmentRequirementB) );

  uint32_t alignmentRequirementC;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        c,
        &descC,
        &alignmentRequirementC) );


  /* ***************************** */

  // Create the Contraction Descriptor
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( handle,
        &desc,
        &descA, a_inds, alignmentRequirementA,
        &descB, b_inds, alignmentRequirementB,
        &descC, c_inds, alignmentRequirementC,
        &descC, c_inds, alignmentRequirementC,
        typeCompute) );

  /* ***************************** */

  // Set the algorithm to use
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
        handle, &find,
        HIPTENSOR_ALGO_DEFAULT) );


  /* ***************************** */

  // Query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(handle,
        &desc,
        &find,
        CUTENSOR_WORKSPACE_RECOMMENDED, &worksize ) );

  // Allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) ) // This is optional!
    {
      work = nullptr;
      worksize = 0;
    }
  }


  /* ***************************** */

  // Create Contraction Plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(handle,
        &plan,
        &desc,
        &find,
        worksize) );

  /* ***************************** */

  hiptensorStatus_t err;

  // Execute the tensor contraction
  err = hiptensorContraction(handle,
      &plan,
      (void*)&alpha, a,
      b,
      (void*)&beta,  c,
      c,
      work, worksize, 0 /* stream */);
  /* hipDeviceSynchronize(); */

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  if ( work ) hipFree( work );

  return 0;
}

extern "C"
int cutensor_contract_c64(hiptensorHandle_t* handle,
    std::complex<float>* c, int c_rank, long* c_dims, int* c_inds,
    std::complex<float>* a, int a_rank, long* a_dims, int* a_inds,
    std::complex<float>* b, int b_rank, long* b_dims, int* b_inds,
    std::complex<float> *alpha,
    std::complex<float> *beta){

  hipDataType typeA = HIP_C_32F;
  hipDataType typeB = HIP_C_32F;
  hipDataType typeC = HIP_C_32F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

  /* ***************************** */

  // Create Tensor Descriptors
  hiptensorTensorDescriptor_t descA;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descA,
        a_rank,
        a_dims,
        NULL,/*stride*/
        typeA, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descB;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descB,
        b_rank,
        b_dims,
        NULL,/*stride*/
        typeB, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descC,
        c_rank,
        c_dims,
        NULL,/*stride*/
        typeC, HIPTENSOR_OP_IDENTITY ) );

  /* ***************************** */

  //Retrieve the memory alignment for each tensor
  uint32_t alignmentRequirementA;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        a,
        &descA,
        &alignmentRequirementA) );

  uint32_t alignmentRequirementB;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        b,
        &descB,
        &alignmentRequirementB) );

  uint32_t alignmentRequirementC;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        c,
        &descC,
        &alignmentRequirementC) );


  /* ***************************** */

  // Create the Contraction Descriptor
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( handle,
        &desc,
        &descA, a_inds, alignmentRequirementA,
        &descB, b_inds, alignmentRequirementB,
        &descC, c_inds, alignmentRequirementC,
        &descC, c_inds, alignmentRequirementC,
        typeCompute) );

  /* ***************************** */

  // Set the algorithm to use
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
        handle, &find,
        HIPTENSOR_ALGO_DEFAULT) );


  /* ***************************** */

  // Query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(handle,
        &desc,
        &find,
        CUTENSOR_WORKSPACE_RECOMMENDED, &worksize ) );

  // Allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) ) // This is optional!
    {
      work = nullptr;
      worksize = 0;
    }
  }


  /* ***************************** */

  // Create Contraction Plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(handle,
        &plan,
        &desc,
        &find,
        worksize) );

  /* ***************************** */

  hiptensorStatus_t err;

  // Execute the tensor contraction
  err = hiptensorContraction(handle,
      &plan,
      alpha, a,
      b,
      beta,  c,
      c,
      work, worksize, 0 /* stream */);
  /* hipDeviceSynchronize(); */

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  if ( work ) hipFree( work );

  return 0;
}

extern "C"
int cutensor_contract_c128(hiptensorHandle_t* handle,
    std::complex<double>* c, int c_rank, long* c_dims, int* c_inds,
    std::complex<double>* a, int a_rank, long* a_dims, int* a_inds,
    std::complex<double>* b, int b_rank, long* b_dims, int* b_inds,
    std::complex<double> *alpha,
    std::complex<double> *beta){

  hipDataType typeA = HIP_C_64F;
  hipDataType typeB = HIP_C_64F;
  hipDataType typeC = HIP_C_64F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_64F;

  /* ***************************** */

  // Create Tensor Descriptors
  hiptensorTensorDescriptor_t descA;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descA,
        a_rank,
        a_dims,
        NULL,/*stride*/
        typeA, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descB;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descB,
        b_rank,
        b_dims,
        NULL,/*stride*/
        typeB, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( handle,
        &descC,
        c_rank,
        c_dims,
        NULL,/*stride*/
        typeC, HIPTENSOR_OP_IDENTITY ) );

  /* ***************************** */

  //Retrieve the memory alignment for each tensor
  uint32_t alignmentRequirementA;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        a,
        &descA,
        &alignmentRequirementA) );

  uint32_t alignmentRequirementB;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        b,
        &descB,
        &alignmentRequirementB) );

  uint32_t alignmentRequirementC;
  HANDLE_ERROR( cutensorGetAlignmentRequirement( handle,
        c,
        &descC,
        &alignmentRequirementC) );


  /* ***************************** */

  // Create the Contraction Descriptor
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( handle,
        &desc,
        &descA, a_inds, alignmentRequirementA,
        &descB, b_inds, alignmentRequirementB,
        &descC, c_inds, alignmentRequirementC,
        &descC, c_inds, alignmentRequirementC,
        typeCompute) );

  /* ***************************** */

  // Set the algorithm to use
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
        handle, &find,
        HIPTENSOR_ALGO_DEFAULT) );


  /* ***************************** */

  // Query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(handle,
        &desc,
        &find,
        CUTENSOR_WORKSPACE_RECOMMENDED, &worksize ) );

  // Allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
    if( hipSuccess != hipMalloc(&work, worksize) ) // This is optional!
    {
      work = nullptr;
      worksize = 0;
    }
  }


  /* ***************************** */

  // Create Contraction Plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(handle,
        &plan,
        &desc,
        &find,
        worksize) );

  /* ***************************** */

  hiptensorStatus_t err;

  // Execute the tensor contraction
  err = hiptensorContraction(handle,
      &plan,
      alpha, a,
      b,
      beta,  c,
      c,
      work, worksize, 0 /* stream */);
  /* hipDeviceSynchronize(); */

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
    printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  if ( work ) hipFree( work );

  return 0;
}

