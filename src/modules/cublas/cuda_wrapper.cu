
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipblas.h>

extern "C" int cublassetstream_wrapper( hipblasHandle_t handle, hipStream_t *stream )
{
    int error;
    if ( stream )
    {
        error = hipblasSetStream(handle, *stream);
    } else
    {
        error = hipblasSetStream(handle, NULL);
    }
    return error;
}
