#include "hip/hip_runtime.h"
#include <complex>
#include "cuda_tailored_permute_kernels.h"

template <typename T>
__global__ void cuda_tensor_permute_1_4_3_2_kernel(T *src_tensor, T *dst_tensor, int *src_dims, unsigned int elements_per_thread){

  __shared__ uint3 sdim, qrs;
  __shared__ unsigned int np;

  unsigned int soffset, doffset, idx;

  idx = blockIdx.x*blockDim.x + threadIdx.x;

  if ( threadIdx.x == 0 ) {
    np = src_dims[0];
    sdim.x = src_dims[1]; sdim.y = src_dims[2]; sdim.z = src_dims[3];
  }
  __syncthreads();

  soffset = blockIdx.y*np;

  if ( threadIdx.x == 0 ) {
    // x == q, y == r, z == s
    // qrs.x + sdims.x*qrs.y + sdims.x*sdims.y*qrs.z = blockIdx.y
    qrs.x = blockIdx.y%sdim.x;
    qrs.y = (blockIdx.y/sdim.x)%sdim.y;
    qrs.z = (blockIdx.y/(sdim.x*sdim.y))%sdim.z;
  }
  __syncthreads();

  doffset = (qrs.x*sdim.y*sdim.z + qrs.y*sdim.z + qrs.z)*np;

  if ( idx < np ) {
    dst_tensor[doffset + idx]= src_tensor[soffset + idx];
  }
}

template __global__ void cuda_tensor_permute_1_4_3_2_kernel<float>(float*, float*, int*, unsigned int);
template __global__ void cuda_tensor_permute_1_4_3_2_kernel<double>(double*, double*, int*, unsigned int);
template __global__ void cuda_tensor_permute_1_4_3_2_kernel<std::complex<float>>(std::complex<float>*, std::complex<float>*, int*, unsigned int);
template __global__ void cuda_tensor_permute_1_4_3_2_kernel<std::complex<double>>(std::complex<double>*, std::complex<double>*, int*, unsigned int);

template <typename T>
__global__ void cuda_tensor_permute_large_first_kernel(T *src_tensor, T *dst_tensor, int *src_dims, int *perm,
    unsigned int order, unsigned int elements_per_thread){

  extern __shared__ unsigned int shared[]; // 4*order*sizeof(unsigned int)

  unsigned int *dims = shared;
  unsigned int *lookup = &dims[order];
  unsigned int *strides = &lookup[order];
  unsigned int *qrs = &strides[order];

  unsigned int soffset, doffset, firstidx, stride;

  if ( threadIdx.x < order ) dims[threadIdx.x] = src_dims[threadIdx.x];
  if ( threadIdx.x >= order && threadIdx.x < 2*order ) lookup[threadIdx.x-order] = perm[threadIdx.x-order];

  firstidx = blockIdx.x*blockDim.x + threadIdx.x;
  __syncthreads();

  if ( threadIdx.x == 0 ) {
    strides[0] = 1;
    for (unsigned int idx=0; idx < order-1; idx++) {
      strides[idx+1] = strides[idx]*dims[idx];
    }
  }
  soffset = blockIdx.y*dims[0];
  __syncthreads();

  if ( threadIdx.x > 0 && threadIdx.x < order ) {
    qrs[threadIdx.x] = (soffset/strides[threadIdx.x])%dims[threadIdx.x];
  }
  __syncthreads();

  doffset = 0;
  stride = dims[0];
  for (unsigned int idx=1; idx < order; idx++) {
    doffset += stride*qrs[lookup[idx]];
    stride *= dims[lookup[idx]];
  }

  if ( firstidx < dims[0] ) {
    dst_tensor[doffset + firstidx]= src_tensor[soffset + firstidx];
  }
}

template __global__ void cuda_tensor_permute_large_first_kernel<float>(float*, float*, int*, int*, unsigned int, unsigned int);
template __global__ void cuda_tensor_permute_large_first_kernel<double>(double*, double*, int*, int*, unsigned int, unsigned int);
template __global__ void cuda_tensor_permute_large_first_kernel<std::complex<float>>(std::complex<float>*, std::complex<float>*, int*, int*, unsigned int, unsigned int);
template __global__ void cuda_tensor_permute_large_first_kernel<std::complex<double>>(std::complex<double>*, std::complex<double>*, int*, int*, unsigned int, unsigned int);

