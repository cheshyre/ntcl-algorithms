#include <complex>

#include <cuda_common.h>
#include "cuda_tailored_permute_kernels.h"

extern "C" void cuda_tensor_permute_1_4_3_2_real32_execute(float *d_src, float *d_dst, int64_t nElems, int np, int* d_sdim, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_1_4_3_2_kernel<float><<<blockGrid, threads_per_block, 0, *stream>>>(d_src, d_dst, d_sdim, elements_per_thread);
  } else {
      cuda_tensor_permute_1_4_3_2_kernel<float><<<blockGrid, threads_per_block>>>(d_src, d_dst, d_sdim, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_1_4_3_2_real64_execute(double *d_src, double *d_dst, int64_t nElems, int np, int* d_sdim, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_1_4_3_2_kernel<double><<<blockGrid, threads_per_block, 0, *stream>>>(d_src, d_dst, d_sdim, elements_per_thread);
  } else {
      cuda_tensor_permute_1_4_3_2_kernel<double><<<blockGrid, threads_per_block>>>(d_src, d_dst, d_sdim, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );

}

extern "C" void cuda_tensor_permute_1_4_3_2_complex64_execute(std::complex<float> *d_src, std::complex<float> *d_dst, int64_t nElems, int np,
    int* d_sdim, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_1_4_3_2_kernel<std::complex<float> ><<<blockGrid, threads_per_block, 0, *stream>>>(d_src, d_dst, d_sdim, elements_per_thread);
  } else {
      cuda_tensor_permute_1_4_3_2_kernel<std::complex<float> ><<<blockGrid, threads_per_block>>>(d_src, d_dst, d_sdim, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );

}

extern "C" void cuda_tensor_permute_1_4_3_2_complex128_execute(std::complex<double> *d_src, std::complex<double> *d_dst, int64_t nElems, int np,
    int* d_sdim, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_1_4_3_2_kernel<std::complex<double> ><<<blockGrid, threads_per_block, 0, *stream>>>(d_src, d_dst, d_sdim, elements_per_thread);
  } else {
      cuda_tensor_permute_1_4_3_2_kernel<std::complex<double> ><<<blockGrid, threads_per_block>>>(d_src, d_dst, d_sdim, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );

}

// Large first unpermuted dimension
extern "C" void cuda_tensor_permute_large_first_real32_execute(float *d_src, float *d_dst, int64_t nElems, int np, int* d_sdim, int* d_perm, int order, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;
  unsigned int bytes_of_shared_memory = 4*order*sizeof(unsigned int);

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_large_first_kernel<float><<<blockGrid, threads_per_block, bytes_of_shared_memory, *stream>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  } else {
      cuda_tensor_permute_large_first_kernel<float><<<blockGrid, threads_per_block, bytes_of_shared_memory>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_large_first_real64_execute(double *d_src, double *d_dst, int64_t nElems, int np, int* d_sdim, int* d_perm, int order, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;
  unsigned int bytes_of_shared_memory = 4*order*sizeof(unsigned int);

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_large_first_kernel<double><<<blockGrid, threads_per_block, bytes_of_shared_memory, *stream>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  } else {
      cuda_tensor_permute_large_first_kernel<double><<<blockGrid, threads_per_block, bytes_of_shared_memory>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_large_first_complex64_execute(std::complex<float> *d_src, std::complex<float> *d_dst, int64_t nElems, int np, int* d_sdim, int* d_perm, int order, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;
  unsigned int bytes_of_shared_memory = 4*order*sizeof(unsigned int);

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_large_first_kernel<std::complex<float>><<<blockGrid, threads_per_block, bytes_of_shared_memory, *stream>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  } else {
      cuda_tensor_permute_large_first_kernel<std::complex<float>><<<blockGrid, threads_per_block, bytes_of_shared_memory>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_large_first_complex128_execute(std::complex<double> *d_src, std::complex<double> *d_dst, int64_t nElems, int np, int* d_sdim, int* d_perm, int order, hipStream_t *stream){

  unsigned int elements_per_thread = 1; // Not implemented
  unsigned int threads_per_block = 256;
  unsigned int elements_per_block = elements_per_thread*threads_per_block;
  unsigned int nBlocksx = ((np-1)/elements_per_block) + 1;
  unsigned int nBlocksy = nElems/np;
  unsigned int bytes_of_shared_memory = 4*order*sizeof(unsigned int);

  dim3 blockGrid(nBlocksx, nBlocksy);
  if ( stream ) {
      cuda_tensor_permute_large_first_kernel<std::complex<double>><<<blockGrid, threads_per_block, bytes_of_shared_memory, *stream>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  } else {
      cuda_tensor_permute_large_first_kernel<std::complex<double>><<<blockGrid, threads_per_block, bytes_of_shared_memory>>>(d_src,
          d_dst, d_sdim, d_perm, order, elements_per_thread);
  }
  check_cuda_error( hipPeekAtLastError() );
}
