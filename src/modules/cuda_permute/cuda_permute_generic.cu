#include <complex>
#include <stdio.h>

#include <cuda_common.h>
#include "cuda_generic_permute_kernel.h"

extern "C" void cuda_tensor_permute_generic_real32_execute(float *d_src, float *d_dst, int64_t nElems, int* d_sdim, int* d_perm, int order){

  // For a TILE_DIM*TILE_DIM block of tensor elements
  // launch TILE_DIM*threadStride number of threads.
  // Each thread is responsible for TILE_DIM/threadStride
  // number of elements, so this number must divide evenly
  int TILE_DIM = 32;
  int threadStride = 8;
  int nBlocks = ((nElems-1) / TILE_DIM/TILE_DIM) + 1;

  dim3 blockGrid(nBlocks);
  dim3 threadsPerBlock(TILE_DIM, threadStride, 1);
  cuda_tensor_permute_generic_kernel<float><<<blockGrid, threadsPerBlock>>>(d_src, d_dst, d_sdim, d_perm, order, TILE_DIM, threadStride);
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_generic_real64_execute(double *d_src, double *d_dst, int64_t nElems, int* d_sdim, int* d_perm, int order){

  // For a TILE_DIM*TILE_DIM block of tensor elements
  // launch TILE_DIM*threadStride number of threads.
  // Each thread is responsible for TILE_DIM/threadStride
  // number of elements, so this number must divide evenly
  int TILE_DIM = 32;
  int threadStride = 8;
  int nBlocks = ((nElems-1) / TILE_DIM/TILE_DIM) + 1;

  dim3 blockGrid(nBlocks);
  dim3 threadsPerBlock(TILE_DIM, threadStride, 1);
  cuda_tensor_permute_generic_kernel<double><<<blockGrid, threadsPerBlock>>>(d_src, d_dst, d_sdim, d_perm, order, TILE_DIM, threadStride);
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_generic_complex64_execute(std::complex<float> *d_src, std::complex<float> *d_dst,
        int64_t nElems, int* d_sdim, int* d_perm, int order){

  // For a TILE_DIM*TILE_DIM block of tensor elements
  // launch TILE_DIM*threadStride number of threads.
  // Each thread is responsible for TILE_DIM/threadStride
  // number of elements, so this number must divide evenly
  int TILE_DIM = 32;
  int threadStride = 8;
  int nBlocks = ((nElems-1) / TILE_DIM/TILE_DIM) + 1;

  dim3 blockGrid(nBlocks);
  dim3 threadsPerBlock(TILE_DIM, threadStride, 1);
  cuda_tensor_permute_generic_kernel<std::complex<float>><<<blockGrid, threadsPerBlock>>>(d_src, d_dst,
              d_sdim, d_perm, order, TILE_DIM, threadStride);
  check_cuda_error( hipPeekAtLastError() );
}

extern "C" void cuda_tensor_permute_generic_complex128_execute(std::complex<double> *d_src, std::complex<double> *d_dst,
        int64_t nElems, int* d_sdim, int* d_perm, int order){

  // For a TILE_DIM*TILE_DIM block of tensor elements
  // launch TILE_DIM*threadStride number of threads.
  // Each thread is responsible for TILE_DIM/threadStride
  // number of elements, so this number must divide evenly
  int TILE_DIM = 32;
  int threadStride = 8;
  int nBlocks = ((nElems-1) / TILE_DIM/TILE_DIM) + 1;

  dim3 blockGrid(nBlocks);
  dim3 threadsPerBlock(TILE_DIM, threadStride, 1);
  cuda_tensor_permute_generic_kernel<std::complex<double>><<<blockGrid, threadsPerBlock>>>(d_src, d_dst,
              d_sdim, d_perm, order, TILE_DIM, threadStride);
  check_cuda_error( hipPeekAtLastError() );
}
